#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <math.h>
#include <string.h>


// Radix sort kernel
__global__ void radix_sort_kernel(const int* d_number_array, int* d_digit_array, int n, int divisor, int lock){
    __shared__ int shared_d_sub_number_array[1024];

    shared_d_sub_number_array[threadIdx.y * 32 + threadIdx.x] = 0;

    int index = blockIdx.x * 1024 + threadIdx.y * 32 + threadIdx.x;
    if (index < n){
        shared_d_sub_number_array[index % 1024] = d_number_array[index];
    }

    __shared__ int shared_d_digit_array[10];
    if (threadIdx.y == 0 && threadIdx.x < 10){
        shared_d_digit_array[threadIdx.x] = 0;
    }
    __syncthreads();

    if (threadIdx.y == 0 && threadIdx.x == 0){
        for (int i = 0; i < 1024; i++){
            if ((blockIdx.x * 1024 + i) >= n){
                break;
            }
            shared_d_digit_array[(shared_d_sub_number_array[i] / divisor) % 10]++;
        }

        for (int i = 1; i < 10; i++){
            shared_d_digit_array[i] += shared_d_digit_array[i-1];
        }

    }
    __syncthreads();

    for (int selected_block = 0; selected_block < gridDim.x; selected_block++){
        if (blockIdx.x == selected_block){
            lock = 1;
            if (threadIdx.y == 0 && threadIdx.x < 10){
                d_digit_array[threadIdx.x] += shared_d_digit_array[threadIdx.x];
            }
            lock = 0;
        } else {
            clock_t start = clock64();
            clock_t now;
            for (;;){
                now = clock64();
                clock_t cycles = now > start ? now - start : now + (0xffffffff - start);
                if (cycles >= 1000000){
                    break;
                }
            }

            for(;;){
                if (!lock){
                    break;
                }
            }
        }

        __syncthreads();
    }

}
__host__ void write_output(char* filename, int* h_number_array, int n){
    FILE * foutput = fopen(filename, "w");
    for (int i = 0; i < n; i++){
        fprintf(foutput, "%d\n", h_number_array[i]);
    }

    fclose(foutput);
}

__host__ void rng(int* arr, int n){
    int seed = 13516118;
    srand(seed);
    for (int i = 0; i < n; i++){
        arr[i] = (int) rand();
    }
}

int main(int argc, char** argv) {
    if (argc != 3){
        printf("Usage: %s number_of_elements file_output_name\n", argv[0]);
        exit(EXIT_FAILURE);
    }

    int n = atoi(argv[1]);
    if (!(n)){
        fprintf(stderr,"parameter is zero or not a number.");
        exit(EXIT_FAILURE);
    }

    int* h_number_array, *h_changed_number_array, *d_number_array, *h_digit_array, *d_digit_array;
    h_number_array = (int*) malloc (n * sizeof(int));
    if (h_number_array == NULL){
        fprintf(stderr, "Cannot allocate host memory.");
        exit(EXIT_FAILURE);
    }
    h_changed_number_array = (int *) malloc (n * sizeof(int));
    if (h_changed_number_array == NULL){
        fprintf(stderr, "Cannot allocate host memory.");
        exit(EXIT_FAILURE);
    }
    hipMalloc((void **) &d_number_array, n * sizeof(int));
    if (d_number_array == NULL){
        fprintf(stderr, "Cannot allocate device memory.");
        exit(EXIT_FAILURE);
    }
    hipMemset(d_number_array, 0, n * sizeof(int));
    hipMalloc((void **) &d_digit_array, 10 * sizeof(int));
    if (d_digit_array == NULL){
        fprintf(stderr, "Cannot allocate device memory.");
        exit(EXIT_FAILURE);
    }
    hipMemset(d_digit_array, 0, 10 * sizeof(int));
    h_digit_array = (int *) malloc (10 * sizeof(int));
    if (h_digit_array == NULL){
        fprintf(stderr, "Cannot allocate device memory.");
        exit(EXIT_FAILURE);
    }

    rng(h_number_array,n);
    hipMemcpy(d_number_array, h_number_array, n * sizeof(int), hipMemcpyHostToDevice);

    int max = INT_MIN;
    for (int i = 0; i < n; i++){
        if (max < h_number_array[i]){
            max = h_number_array[i];
        }
    }

    dim3 grid, block;
    block.x = 32;
    block.y = 32;
    grid.x = n / (block.x * block.y);
    if (n % (block.x * block.y) != 0)
        grid.x++;

    int lock = 0;
    for (int divisor = 1; max/divisor > 0; divisor *= 10){
        radix_sort_kernel<<<grid, block>>>(d_number_array, d_digit_array, n, divisor, lock);
        hipDeviceSynchronize();

        hipMemcpy(h_digit_array, d_digit_array, 10 * sizeof(int), hipMemcpyDeviceToHost);
        for (int i = n - 1; i >= 0; i--){
            h_changed_number_array[h_digit_array[(h_number_array[i] / divisor) % 10] - 1] = h_number_array[i];

            h_digit_array[(h_number_array[i] / divisor) % 10]--;
        }

        memcpy(h_number_array, h_changed_number_array, n * sizeof(int));
        hipMemcpy(d_number_array, h_changed_number_array, n * sizeof(int), hipMemcpyHostToDevice);
        hipMemset(d_digit_array, 0, 10 * sizeof(int));
    }

    write_output(argv[2], h_number_array, n);

    free(h_number_array);
    free(h_changed_number_array);
    free(h_digit_array);
    hipFree(d_number_array);
    hipFree(d_digit_array);
    return 0;
}